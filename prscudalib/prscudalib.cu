#include "hip/hip_runtime.h"

#ifndef _PRS_CUDA_FUNC_
#define _PRS_CUDA_FUNC_

#include <dsp\bitbuf.h>
#include <vector>
#include "hip/hip_runtime.h"
#include ""

#define imin(a,b) (a<b?a:b)

//__constant__ int seqs[16384]; // ������������ ����� ����������� ������ �� ���� GTX750 = 64k

__global__ void akf_kernel(int *akf, const int *seq1, const int *seq2, const int seq_size) { // ������ ����� ���� �� (�� ���� �������)
    //__shared__ int cache[THREADS_PER_BLOCK];
    extern __shared__ int cache[];
    int tid = threadIdx.x + blockIdx.y * blockDim.x; // 1-� ����������� ������ ����������� 1 �������� ��
    int cacheIndex = threadIdx.x;
    //int temp = seq1[tid] ^ seq2[(blockIdx.x + tid) % seq_size];
    int temp = seq1[tid] * seq2[(blockIdx.x + tid) % seq_size]; 
    cache[cacheIndex] = temp;
    __syncthreads();
    temp = blockDim.x / 2;
    while (temp != 0) {
        if (cacheIndex < temp)
            cache[cacheIndex] += cache[cacheIndex + temp];
        __syncthreads();
        temp /= 2;
    }
    if (cacheIndex == 0) // � 0��� �������� ����� ���� ��������� �������
        akf[blockIdx.x * gridDim.y + blockIdx.y] = cache[0]; // 2-� ����������� ������ ����������� ������
}

namespace dsp {
    namespace prs {
        int cudaXcorr(std::vector<float> &akf,
            const dsp::BitBuffer<dsp::u32>  &seq1,
            const dsp::BitBuffer<dsp::u32>  &seq2,
            const bool                      normalize = true) {

            if (seq1.size() != seq2.size())
                return 128; // ������, ������������������ ������ �����

            int *dev_seq1 = 0;
            int *dev_seq2 = 0;
            int *dev_akf = 0;
            unsigned int threads_per_block = imin(1024, seq1.size());
            unsigned int blocks_per_seq = imin(65535, (seq1.size() + threads_per_block - 1) / threads_per_block);
            hipError_t err;

            int *int_seq1 = new int[seq1.size()];
            int *int_seq2 = new int[seq1.size()];
            int *akf_temp = new int[seq1.size() * blocks_per_seq];

            for (int k = 0; k < seq1.size(); ++k) {
                //int_seq1[k] = seq1[k];
                //int_seq2[k] = seq2[k];
                if (seq1[k] == 1)
                    int_seq1[k] = 1;
                else
                    int_seq1[k] = -1;
                if (seq2[k] == 1)
                    int_seq2[k] = 1;
                else
                    int_seq2[k] = -1;
            }
            // GPU part.
            err = hipMalloc((void**)&dev_seq1, seq1.size() * sizeof(int));
            //err = hipMemcpyToSymbol(HIP_SYMBOL(seqs), int_seq1, seq1.size() * sizeof(int));
            if (err != hipSuccess) return (int)err;
            err = hipMalloc((void**)&dev_seq2, seq1.size() * sizeof(int));
            if (err != hipSuccess) return (int)err;
            err = hipMalloc((void**)&dev_akf, seq1.size() * blocks_per_seq * sizeof(int));
            if (err != hipSuccess) return (int)err;

            err = hipMemcpy(dev_seq1, int_seq1, seq1.size() * sizeof(int), hipMemcpyHostToDevice);
            if (err != hipSuccess) return (int)err;
            err = hipMemcpy(dev_seq2, int_seq2, seq1.size() * sizeof(int), hipMemcpyHostToDevice);
            if (err != hipSuccess) return (int)err;

            dim3 grids(seq1.size(), blocks_per_seq);
            dim3 threads(threads_per_block);
            akf_kernel <<< grids, threads, threads_per_block * sizeof(int) >>> (dev_akf, dev_seq1, dev_seq2, seq1.size());
            err = hipGetLastError();
            if (err != hipSuccess) return (int)err;

            err = hipMemcpy(akf_temp, dev_akf, seq1.size() * blocks_per_seq * sizeof(int), hipMemcpyDeviceToHost);
            if (err != hipSuccess) return (int)err;

            for (int i = 0; i < seq1.size(); ++i) { // ������������ �������� �� �� ������ � ���� �����
                akf[i] = 0;
                for (int k = 0; k < blocks_per_seq; ++k)
                    akf[i] += akf_temp[i*blocks_per_seq + k];
            }

            if (normalize)
                for (int i = 0; i < seq1.size(); ++i)
                    akf[i] /= seq1.size();

            hipFree(dev_akf);
            hipFree(dev_seq1);
            hipFree(dev_seq2);

            delete[] int_seq1;
            delete[] int_seq2;
            delete[] akf_temp;

            return 0; // ��� ������
        };
    }
}

#endif

