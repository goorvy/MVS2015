#include "hip/hip_runtime.h"

//#include <windows.h>
//#include <dsp\filebuf.h>
#include <conio.h>
#include <ctime>
#include <stdio.h>
#include <vector>
#include <dsp\bitbuf.h>
#include "C:\Users\dshubin\Documents\Visual Studio 2008\Projects\deBruijnFile\deBruijnFile\prs_debruijn_seqs.h"
#include <clocale>
#include "hip/hip_runtime.h"
#include ""

//BOOL CALLBACK EnumWndProc(HWND hwnd, LPARAM lParam) { // ��� ��������� ����������� ���� �������
//    if (GetWindowThreadProcessId(hwnd, NULL) == GetCurrentThreadId()) {
//        *(HWND*)lParam = hwnd;
//        return FALSE;
//    }
//    return TRUE;
//}
//
//#ifndef MAX_PATH
//#define MAX_PATH 260
//#endif

#define imin(a,b) (a<b?a:b)

__global__ void akf_kernel(int *akf, const int *seq1, const int *seq2, const int seq_size) { // ������ ����� ���� �� (�� ���� �������)
                                                                                             //__shared__ int cache[THREADS_PER_BLOCK];
    extern __shared__ int cache[];
    int tid = threadIdx.x + blockIdx.y * blockDim.x; // 1-� ����������� ������ ����������� 1 �������� ��
    int cacheIndex = threadIdx.x;
    int temp = seq1[tid] * seq2[(blockIdx.x + tid) % seq_size];
    cache[cacheIndex] = temp;
    __syncthreads();
    temp = blockDim.x / 2;
    while (temp != 0) {
        if (cacheIndex < temp)
            cache[cacheIndex] += cache[cacheIndex + temp];
        __syncthreads();
        temp /= 2;
    }
    if (cacheIndex == 0) // � 0��� �������� ����� ���� ��������� �������
        akf[blockIdx.x * gridDim.y + blockIdx.y] = cache[0]; // 2-� ����������� ������ ����������� ������
}

int main()
{
    int seq_length;
    int seq_index;
    int seqs_amount;
    clock_t t_start, t_end;
    std::vector<float> akf;

    std::setlocale(LC_CTYPE, "Russian_Russia.1251");

    printf("���������� ��� ����� ������������������ �� ������ �� GPU\n");

    printf("������� ����� ������������������ �� ������ = ");
    scanf("%d", &seq_length);

    printf("������� ��������� ������ ������������������� �� ������ = ");
    scanf("%d", &seq_index);

    printf("������� ���������� ������������������� �� ������ = ");
    scanf("%d", &seqs_amount);

    //OPENFILENAME OpenFileName; // ��������� ��� �������
    //static TCHAR openfilename[255]; // ����� ����� �����
    //HWND hWnd; // ���������� ����
    //           // ����� ����� ����� ��� ����������
    //EnumWindows(EnumWndProc, (LPARAM)&hWnd); // ��������� ����������� ����������� ����
    //ZeroMemory(&OpenFileName, sizeof(OPENFILENAME));
    //OpenFileName.lStructSize = OPENFILENAME_SIZE_VERSION_400A;
    //OpenFileName.hwndOwner = hWnd;
    //OpenFileName.lpstrFile = openfilename;
    //OpenFileName.nMaxFile = MAX_PATH;
    //OpenFileName.lpstrFilter = "Binary Files\0*.bin\0\0"; // ������ ���� ������ � �������
    //OpenFileName.nFilterIndex = 1;
    //OpenFileName.lpstrFileTitle = NULL;
    //OpenFileName.nMaxFileTitle = 0;
    //OpenFileName.lpstrInitialDir = "C:\\Users\\dshubin\\Documents\\�����\\��� �� ������\\Save";
    //OpenFileName.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST | OFN_NOCHANGEDIR;

    //if (!GetOpenFileName(&OpenFileName)) { // ����� ������� ������ ����� ��� ���������� �����
    //    cout << "������ �� ����� ��������.\n"; // � ������� ������ "������" �������������� ������������ ��������� �� ������ � ����
    //    return 1;
    //}
    //dsp::u32IFBB ifb(OpenFileName.lpstrFile);
    //u32* buf = new u32[SEQ_LENGTH];
    //for (int i = 0; i < SEQ_LENGTH; ++i)
    //    buf[i] = 0;
    //ifb(buf, SEQ_LENGTH);
    //dsp::BitBuffer<dsp::u32> bin_seq(buf, SEQ_LENGTH);

    akf.resize(seq_length);
    dsp::BitBuffer<dsp::u32> bin_seq1(seq_length);
    dsp::BitBuffer<dsp::u32> bin_seq2(seq_length);

    //printf("Sequence length = %d\nSequence index = %d\n", seq_length, seq_index);

    t_start = clock();
    dsp::prs::PRSDeBruijnSeq debruijn_seqs(seq_length);
    bin_seq1 = debruijn_seqs.get_seqs(seq_index);
    t_end = clock();

    printf("������������������ �� ������ ����� %d � �������� %d\n������������ �� %f ������.\n",
        seq_length, seq_index, ((float)t_end - (float)t_start) / CLOCKS_PER_SEC);


        int *dev_seq1 = 0;
        int *dev_seq2 = 0;
        int *dev_akf = 0;
        unsigned int threads_per_block = imin(1024, seq1.size());
        unsigned int blocks_per_seq = imin(65535, (seq1.size() + threads_per_block - 1) / threads_per_block);
        hipError_t err;

        int *int_seq1 = new int[seq1.size()];
        int *int_seq2 = new int[seq1.size()];
        int *akf_temp = new int[seq1.size() * blocks_per_seq];

        for (int k = 0; k < seq1.size(); ++k) {
            if (seq1[k] == 1)
                int_seq1[k] = 1;
            else
                int_seq1[k] = -1;
            if (seq2[k] == 1)
                int_seq2[k] = 1;
            else
                int_seq2[k] = -1;
        }
        // GPU part.
        err = hipMalloc((void**)&dev_seq1, seq1.size() * sizeof(int));
        //err = hipMemcpyToSymbol(HIP_SYMBOL(seqs), int_seq1, seq1.size() * sizeof(int));
        if (err != hipSuccess) return (int)err;
        err = hipMalloc((void**)&dev_seq2, seq1.size() * sizeof(int));
        if (err != hipSuccess) return (int)err;
        err = hipMalloc((void**)&dev_akf, seq1.size() * blocks_per_seq * sizeof(int));
        if (err != hipSuccess) return (int)err;

        err = hipMemcpy(dev_seq1, int_seq1, seq1.size() * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) return (int)err;
        err = hipMemcpy(dev_seq2, int_seq2, seq1.size() * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) return (int)err;

        dim3 grids(seq1.size(), blocks_per_seq);
        dim3 threads(threads_per_block);
        akf_kernel << < grids, threads, threads_per_block * sizeof(int) >> > (dev_akf, dev_seq1, dev_seq2, seq1.size());
        err = hipGetLastError();
        if (err != hipSuccess) return (int)err;

        err = hipMemcpy(akf_temp, dev_akf, seq1.size() * blocks_per_seq * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) return (int)err;

        for (int i = 0; i < seq1.size(); ++i) { // ������������ �������� �� �� ������ � ���� �����
            akf[i] = 0;
            for (int k = 0; k < blocks_per_seq; ++k)
                akf[i] += akf_temp[i*blocks_per_seq + k];
        }

        hipFree(dev_akf);
        hipFree(dev_seq1);
        hipFree(dev_seq2);

        delete[] int_seq1;
        delete[] int_seq2;
        delete[] akf_temp;

    return 0;
}

